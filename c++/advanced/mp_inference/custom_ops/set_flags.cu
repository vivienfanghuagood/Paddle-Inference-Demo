#include "hip/hip_runtime.h"
#include "paddle/extension.h"

__device__ bool is_in_list(const int64_t id, const int64_t *ids, int bs_id) {
    bool is_in_list = false;
    for (int i = 0; i < bs_id; i++) {
        if (ids[i] == id) {
            return true;
        }
    }
    return is_in_list;
}

__global__ void set_value_by_id(const bool *stop_flags, const int64_t *ids, bool *stop_flags_out, int bs, int bs_id) {
    int tid = threadIdx.x;
    if (tid < bs && !is_in_list(tid, ids, bs_id)) {
        stop_flags_out[tid] = true;
    }
}

std::vector<paddle::Tensor> SetFlags(const paddle::Tensor& stop_flags, const paddle::Tensor& gather_id) {
    PD_CHECK(gather_id.dtype() == paddle::DataType::INT64);
    PD_CHECK(stop_flags.dtype() == paddle::DataType::BOOL);
    auto cu_stream = stop_flags.stream();
    std::vector<int64_t> flag_shape = stop_flags.shape();
    std::vector<int64_t> id_shape = gather_id.shape();
    auto stop_flags_out = stop_flags.copy_to(stop_flags.place(), false); // gpu -> gpu
    if (flag_shape[0] == id_shape[0]) {
        return {stop_flags_out};
    }
    int flag_bs = flag_shape[0];
    int id_bs = id_shape[0];
    int block_size = (flag_bs + 32 - 1) / 32 * 32;
    set_value_by_id<<<1, block_size, 0, cu_stream>>>(stop_flags.data<bool>(), gather_id.data<int64_t>(), stop_flags_out.data<bool>(), flag_bs, id_bs);
    return {stop_flags_out};
}

std::vector<std::vector<int64_t>> SetFlagsInferShape(const std::vector<int64_t>& stop_flags_shape, const std::vector<int64_t>& gather_id_shape) {
    return {stop_flags_shape};
}

std::vector<paddle::DataType> SetFlagsInferDtype(const paddle::DataType& stop_flags_dtype, const paddle::DataType& gather_id_dtype) {
    return {stop_flags_dtype};
}

PD_BUILD_OP(set_flags)
    .Inputs({"stop_flags", "gather_id"})
    .Outputs({"stop_flags_out"})
    .SetKernelFn(PD_KERNEL(SetFlags))
    .SetInferShapeFn(PD_INFER_SHAPE(SetFlagsInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(SetFlagsInferDtype));