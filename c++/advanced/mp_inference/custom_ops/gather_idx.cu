#include "hip/hip_runtime.h"
#include "paddle/extension.h"

template <paddle::DataType D>
class PDTraits;

template <>
class PDTraits<paddle::DataType::FLOAT32> {
public:
  typedef float DataType;
  typedef float data_t;
};

template <>
class PDTraits<paddle::DataType::FLOAT16> {
public:
  typedef half DataType;
  typedef paddle::float16 data_t;
};

template <>
class PDTraits<paddle::DataType::BFLOAT16> {
public:
  typedef __hip_bfloat16 DataType;
  typedef paddle::bfloat16 data_t;
};

template<typename T>
__global__ void get_value_by_id(const T *logits, 
                                const int *ids, 
                                T *logits_out, 
                                int bs, int seq_len, int length) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int idx = bid * blockDim.x + tid;
    for (int i = idx; i < bs * length; i += gridDim.x * blockDim.x) {
        int bi = i / length;
        int lane = i % length;
        int si = ids[bi];
        if (si == -1) {
            si = 0;
        }
        const T *logits_now = logits + bi * seq_len * length + si * length;
        T * logits_out_now = logits_out + bi * length;
        logits_out_now[lane] = logits_now[lane];
    }
}

template <paddle::DataType D>
std::vector<paddle::Tensor> gather_idx(const paddle::Tensor& logits, const paddle::Tensor& gather_id) {
    
    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;

    PD_CHECK(gather_id.dtype() == paddle::DataType::INT32);
    auto cu_stream = logits.stream();
    std::vector<int64_t> logits_shape = logits.shape();
    std::vector<int64_t> id_shape = gather_id.shape();
    int logits_bs = logits_shape[0];
    int seq_len = logits_shape[1];
    int logits_len = logits_shape[2];
    auto logits_out = paddle::full({logits_bs, logits_len}, -1, logits.type(), logits.place());
    int id_bs = id_shape[0];
    int64_t numels = logits_bs * logits_len;
    int block_size = 128;
    int grid_size = (numels + block_size - 1) / block_size;
    get_value_by_id<<<grid_size, block_size, 0, cu_stream>>>(reinterpret_cast<DataType_*>(const_cast<data_t*>(logits.data<data_t>())), 
                                                             gather_id.data<int>(), 
                                                             reinterpret_cast<DataType_*>(const_cast<data_t*>(logits_out.data<data_t>())), 
                                                             logits_bs, seq_len, logits_len);
    return {logits_out};
}

std::vector<paddle::Tensor> GatherIdx(const paddle::Tensor& logits, const paddle::Tensor& gather_id) {
    switch (logits.type()) {
        case paddle::DataType::BFLOAT16: {
            return gather_idx<paddle::DataType::BFLOAT16>(
                logits,
                gather_id
            );
        }
        case paddle::DataType::FLOAT16: {
            return gather_idx<paddle::DataType::FLOAT16>(
                logits,
                gather_id
            );
        }
        case paddle::DataType::FLOAT32: {
            return gather_idx<paddle::DataType::FLOAT32>(
                logits,
                gather_id
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only bfloat16, float16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> GatherIdxInferShape(const std::vector<int64_t>& logits_shape, const std::vector<int64_t>& gather_id_shape) {
    std::vector<int64_t> out_shape = {logits_shape[0], logits_shape[2]};
    return {out_shape};
}

std::vector<paddle::DataType> GatherIdxInferDtype(const paddle::DataType& logits_dtype, const paddle::DataType& gather_id_dtype) {
    return {logits_dtype};
}

PD_BUILD_OP(gather_idx)
    .Inputs({"logits", "gather_id"})
    .Outputs({"logits_out"})
    .SetKernelFn(PD_KERNEL(GatherIdx))
    .SetInferShapeFn(PD_INFER_SHAPE(GatherIdxInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(GatherIdxInferDtype));