#include "hip/hip_runtime.h"
#include "paddle/extension.h"


template <paddle::DataType D>
class PDTraits;

template <>
class PDTraits<paddle::DataType::FLOAT32> {
public:
  typedef float DataType;
  typedef float data_t;
};

template <>
class PDTraits<paddle::DataType::FLOAT16> {
public:
  typedef half DataType;
  typedef paddle::float16 data_t;
};

template <>
class PDTraits<paddle::DataType::BFLOAT16> {
public:
  typedef __hip_bfloat16 DataType;
  typedef paddle::bfloat16 data_t;
};

template<typename T>
__global__ inline void min_length_logits_process(T* logits,
                                                 const int64_t *cur_len,
                                                 const int64_t *min_len,
                                                 const int64_t *eos_token_id,
                                                 const int64_t bs,
                                                 const int64_t length,
                                                 const int64_t end_length) {
    int bi = blockIdx.x;
    if (cur_len[bi] < 0) {
        return;
    }
    if (cur_len[bi] < min_len[bi]) {
        for (int i=0; i < end_length; i++) {
            logits[bi * length + eos_token_id[i]] = -1e4;
        }
    }
}

template<>
__global__ inline void min_length_logits_process<half>(half* logits,
                                                       const int64_t *cur_len,
                                                       const int64_t *min_len,
                                                       const int64_t *eos_token_id,
                                                       const int64_t bs,
                                                       const int64_t length,
                                                       const int64_t end_length) {
    int bi = blockIdx.x;
    if (cur_len[bi] < 0) {
        return;
    }
    if (cur_len[bi] < min_len[bi]) {
        for (int i=0; i < end_length; i++) {
            logits[bi * length + eos_token_id[i]] = -1e4;
        }
    }
}


__global__ void update_repeat_times(const int64_t *pre_ids, 
                                    const int64_t *cur_len,
                                    int *repeat_times, 
                                    const int64_t bs, 
                                    const int64_t length, 
                                    const int64_t length_id) {
    int bi = blockIdx.x;
    if (cur_len[bi] < 0) {
        return;
    }
    int tid = threadIdx.x;
    const int64_t *pre_ids_now = pre_ids + bi * length_id;
    int *repeat_times_now = repeat_times + bi * length;
    for (int i = tid; i < length_id; i += blockDim.x) {
        int64_t id = pre_ids_now[i];
        if (id < 0) break;
        atomicAdd(&repeat_times_now[id], 1);
    }
}

template<typename T>
__global__ void update_value_by_repeat_times(const int *repeat_times, 
                                             const T *penalty_scores, 
                                             const T *frequency_score, 
                                             const T *presence_score, 
                                             T *logits, 
                                             const int64_t bs, 
                                             const int64_t length) {
    int bi = blockIdx.x;
    int tid = threadIdx.x;
    T *logits_now = logits + bi * length;
    const int *repeat_times_now = repeat_times + bi * length;
    float alpha = static_cast<float>(penalty_scores[bi]);
    float beta = static_cast<float>(frequency_score[bi]);
    float gamma = static_cast<float>(presence_score[bi]);
    for (int i = tid; i < length; i += blockDim.x) {
        int times = repeat_times_now[i];
        // printf("bi: %d, ti: %d, repeat_times: %d\n", bi, tid, times);
        if (times == 0) continue;
        float logit_now = static_cast<float>(logits_now[i]);
        logit_now = logit_now < 0 ? logit_now * alpha : logit_now / alpha;
        logits_now[i] = static_cast<T>(logit_now - times * beta - gamma);
        // printf("bi: %d, ti: %d, repeat_times: %d, presence_score: %f, frequency_score: %f, presence_score: %f, logit_now: %f, logits: %f\n", 
                // bi, tid, times, alpha, beta, gamma, (float)logit_now, (float)logits_now[i]);
    }
}

template <paddle::DataType D>
std::vector<paddle::Tensor> token_penalty_multi_scores_kernel(const paddle::Tensor& pre_ids, 
                                                              const paddle::Tensor& logits, 
                                                              const paddle::Tensor& penalty_scores, 
                                                              const paddle::Tensor& frequency_score, 
                                                              const paddle::Tensor& presence_score,
                                                              const paddle::Tensor& cur_len,
                                                              const paddle::Tensor& min_len,
                                                              const paddle::Tensor& eos_token_id) {

    typedef PDTraits<D> traits_;
    typedef typename traits_::DataType DataType_;
    typedef typename traits_::data_t data_t;
    auto cu_stream = logits.stream();
    std::vector<int64_t> shape = logits.shape();
    auto repeat_times = paddle::full(shape, 0, paddle::DataType::INT32, pre_ids.place());
    int64_t bs = shape[0];
    int64_t length = shape[1];
    int64_t length_id = pre_ids.shape()[1];
    auto logits_out = logits.copy_to(logits.place(), false); // gpu -> gpu

    int64_t end_length = eos_token_id.shape()[0];

    min_length_logits_process<<<bs, 1, 0, cu_stream>>>(
        reinterpret_cast<DataType_*>(const_cast<data_t*>(logits_out.data<data_t>())),
        cur_len.data<int64_t>(),
		min_len.data<int64_t>(),
		eos_token_id.data<int64_t>(),
		bs, length, end_length);

    int block_size_1 = (length_id + 32 - 1) / 32 * 32;
    block_size_1 = min(block_size_1, 512);
    update_repeat_times<<<bs, block_size_1, 0, cu_stream>>>(pre_ids.data<int64_t>(), cur_len.data<int64_t>(), repeat_times.data<int>(), bs, length, length_id);
    int block_size_2 = (length + 32 - 1) / 32 * 32;
    block_size_2 = min(block_size_2, 512);
    update_value_by_repeat_times<DataType_><<<bs, block_size_2, 0, cu_stream>>>(
        repeat_times.data<int>(),
        reinterpret_cast<DataType_*>(const_cast<data_t*>(penalty_scores.data<data_t>())),
        reinterpret_cast<DataType_*>(const_cast<data_t*>(frequency_score.data<data_t>())),
        reinterpret_cast<DataType_*>(const_cast<data_t*>(presence_score.data<data_t>())),
        reinterpret_cast<DataType_*>(const_cast<data_t*>(logits_out.data<data_t>())),
        bs, length);
    return {logits_out};
}

std::vector<paddle::Tensor> TokenPenaltyMultiScores(const paddle::Tensor& pre_ids, 
                                                    const paddle::Tensor& logits, 
                                                    const paddle::Tensor& penalty_scores, 
                                                    const paddle::Tensor& frequency_scores, 
                                                    const paddle::Tensor& presence_scores,
                                                    const paddle::Tensor& cur_len,
                                                    const paddle::Tensor& min_len,
                                                    const paddle::Tensor& eos_token_id) {

    switch (logits.type()) {
        case paddle::DataType::BFLOAT16: {
            return token_penalty_multi_scores_kernel<paddle::DataType::BFLOAT16>(
                pre_ids,
                logits,
                penalty_scores,
                frequency_scores,
                presence_scores,
                cur_len,
                min_len,
                eos_token_id
            );
        }
        case paddle::DataType::FLOAT16: {
            return token_penalty_multi_scores_kernel<paddle::DataType::FLOAT16>(
                pre_ids,
                logits,
                penalty_scores,
                frequency_scores,
                presence_scores,
                cur_len,
                min_len,
                eos_token_id
            );
        }
        case paddle::DataType::FLOAT32: {
            return token_penalty_multi_scores_kernel<paddle::DataType::FLOAT32>(
                pre_ids,
                logits,
                penalty_scores,
                frequency_scores,
                presence_scores,
                cur_len,
                min_len,
                eos_token_id
            );
        }
        default: {
            PD_THROW(
                "NOT supported data type. "
                "Only float16 and float32 are supported. ");
            break;
        }
    }
}

std::vector<std::vector<int64_t>> TokenPenaltyMultiScoresInferShape(const std::vector<int64_t>& pre_ids_shape, 
                                                                    const std::vector<int64_t>& logits_shape, 
                                                                    const std::vector<int64_t>& penalty_scores_shape, 
                                                                    const std::vector<int64_t>& frequency_scores_shape, 
                                                                    const std::vector<int64_t>& presence_scores_shape,
                                                                    const std::vector<int64_t>& cur_len_shape,
                                                                    const std::vector<int64_t>& min_len_shape,
                                                                    const std::vector<int64_t>& eos_token_id_shape) {
    return {logits_shape};
}

std::vector<paddle::DataType> TokenPenaltyMultiScoresInferDtype(const paddle::DataType& pre_ids_dtype, 
                                                                const paddle::DataType& logits_dtype, 
                                                                const paddle::DataType& penalty_scores_dtype, 
                                                                const paddle::DataType& frequency_scores_dtype, 
                                                                const paddle::DataType& presence_scores_dtype,
                                                                const paddle::DataType& cur_len_dtype,
                                                                const paddle::DataType& min_len_dtype,
                                                                const paddle::DataType& eos_token_id_dtype) {
    return {logits_dtype};
}

PD_BUILD_OP(get_token_penalty_multi_scores)
    .Inputs({"pre_ids", "logits", "penalty_scores", "frequency_scores", "presence_scores", "cur_len", "min_len", "eos_token_id"})
    .Outputs({"logits_out"})
    .SetKernelFn(PD_KERNEL(TokenPenaltyMultiScores))
    .SetInferShapeFn(PD_INFER_SHAPE(TokenPenaltyMultiScoresInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(TokenPenaltyMultiScoresInferDtype));